#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define MEMMAX 30000
#define DEVMEMMAX 32 // 16kB / 512

__global__ void devBrainfuck(char *devCode, char *outMem)
{
  int tid = threadIdx.x;
  int tcnt = blockDim.x;
  char *cp = devCode;
  extern __shared__ char mem[];
  char *mp = &mem[tid];
  int i, kc;

  // 初期化
  mem[tid] = tid;
  for (i = 1; i < tcnt; i++) {
    mem[i * tcnt + tid] = 0;
  }

  // 命令実行
  while (*cp != '}') {
    switch (*cp) {
    case '+':
      (*mp)++;
      break;
    case '-':
      (*mp)--;
      break;
    case '>':
      mp += tcnt;
      break;
    case '<':
      mp -= tcnt;
      break;
    case '[':
      if (*mp == 0) {
	cp++;
	kc = 0;
	while (*cp != ']' || kc > 0) {
	  if (*cp == '[') kc++;
	  if (*cp == ']') kc--;
	  cp++;
	}
      }
      break;
    case ']':
      if (*mp != 0) {
	cp--;
	kc = 0;
	while (*cp != '[' || kc > 0) {
	  if (*cp == ']') kc++;
	  if (*cp == '[') kc--;
	  cp--;
	}
      }
      break;
    }
    cp++;
  } 
  outMem[tid] = *mp;
}

int brainfuck(char *code)
{
  char *cp = code;
  char mem[MEMMAX];
  char *mp = mem;
  int  kc; // カッコカウンタ(カッコワルイ)
  int  tc;
  int  cc;
  char *devCode, *outMem;
  int  i;

  // BFホストメモリ初期化
  for (i = 0; i < MEMMAX; i++) {
    mem[i] = 0;
  }

  // BF命令実行
  while (*cp) {
    switch (*cp) {
    case '+':
      (*mp)++;
      break;
    case '-':
      (*mp)--;
      break;
    case '>':
      mp++;
      break;
    case '<':
      mp--;
      break;
    case '.':
      printf("%c(%d)\n", *mp, *mp);
      //putchar(*mp);
      break;
    case ',':
      while ((*mp = getchar()) == '\n');
      break;
    case '[':
      if (*mp == 0) {
	cp++;
	kc = 0;
	while (*cp != ']' || kc > 0) {
	  if (*cp == '[') kc++;
	  if (*cp == ']') kc--;
	  cp++;
	}
      }
      break;
    case ']':
      if (*mp != 0) {
	cp--;
	kc = 0;
	while (*cp != '[' || kc > 0) {
	  if (*cp == ']') kc++;
	  if (*cp == '[') kc--;
	  cp--;
	}
      }
      break;

    case '{': // 拡張
      tc = *mp; // スレッド数
      cp++;
      cc = 0;
      while (cp[cc++] != '}');
      hipMalloc((void**)&devCode, sizeof(char) * cc); // 命令(デバイス)
      hipMemcpy(devCode, cp, sizeof(char) * cc, hipMemcpyHostToDevice);

      hipMalloc((void**)&outMem,  sizeof(char) * tc); // 実行結果

      //for (i = 0; i < cc; i++) putchar(cp[i]);
      //printf("/cc=%d tc=%d\n", cc, tc);

      // 実行
      dim3 grid(1, 1);
      dim3 block(tc, 1, 1);
      devBrainfuck <<< grid, block, tc * DEVMEMMAX >>> (devCode, outMem);

      // 結果取得
      hipMemcpy(mp+1, outMem, sizeof(char) * tc, hipMemcpyDeviceToHost);
      
      //for (i = 0; i < tc; i++) printf("[%d]%c\n", i, mp[i+1]);

      hipFree(devCode);
      hipFree(outMem);

      cp += cc - 1;
      break;
    }
    cp++;
  }

  return 0;
}

int main(int argc, char *argv)
{
  char *code = "+++++++++[>++++++++>+++++++++++>+++++<<<-]>.>++.+++++++..+++.>-.------------.<++++++++.--------.+++.------.--------.>+.";
  
  brainfuck(code); // Hello World
  printf("\n");
  
  code = ">>+++[<+++++++++>-]<-{>>+++++++++[<++++++++++>-]<+++++++<[>+<-]>}[>.[-]<[->+<]>-]";
  brainfuck(code); // print a to z
}
